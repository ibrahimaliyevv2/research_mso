#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernel.h"


__device__ float tempParticle1[NUM_OF_DIMENSIONS];
__device__ float tempParticle2[NUM_OF_DIMENSIONS];

/* Objective function
0: Levy 3-dimensional
1: Shifted Rastigrin's Function
2: Shifted Rosenbrock's Function
3: Shifted Griewank's Function
4: Shifted Sphere's Function
*/
/**
 * Runs on the GPU, called from the GPU.
*/
__device__ float fitness_function(float x[]) {
    float res = 0;
    float somme = 0;
    float produit = 0;
    float y1 = 0;
    float yn = 0;

    switch (SELECTED_OBJ_FUNC)  {
        case 0: 
            y1 = 1 + (x[0] - 1)/4;
            yn = 1 + (x[NUM_OF_DIMENSIONS-1] - 1)/4;

            res += pow(sin(phi*y1), 2);

            for (int i = 0; i < NUM_OF_DIMENSIONS-1; i++) {
                float y = 1 + (x[i] - 1)/4;
                float yp = 1 + (x[i+1] - 1)/4;
                res += pow(y - 1, 2)*(1 + 10*pow(sin(phi*yp), 2)) + pow(yn - 1, 2);
            }
            break;
        case 1: 
            for (int i = 0; i < NUM_OF_DIMENSIONS; i++) {
                float zi = x[i] - 0;
                res += pow(zi, 2) - 10*cos(2*phi*zi) + 10;
            }
            res -= 330;
            break;
        
        case 2:
            for (int i = 0; i < NUM_OF_DIMENSIONS-1; i++) {
                float zi = x[i] - 0 + 1;
                float zip1 = x[i+1] - 0 + 1;
                res += 100 * ( pow(pow(zi, 2) - zip1, 2)) + pow(zi - 1, 2);
            }
            res += 390;
            break;
        case 3:
            for (int i = 0; i < NUM_OF_DIMENSIONS; i++) {
                float zi = x[i] - 0;
                somme += pow(zi, 2)/4000;
                produit *= cos(zi/pow(i+1, 0.5));
            }
            res = somme - produit + 1 - 200; 
            break;
        case 4:
            for(int i = 0; i < NUM_OF_DIMENSIONS; i++) {
                float zi = x[i] - 0;
                res += pow(zi, 2);
            }
            res -= 450;
            break;
    }

    return res;
}

/**
 * 
 * Runs on the GPU, called from the CPU or the GPU
*/
__global__ void kernelUpdateParticle(float *positions, float *velocities, 
                                     float *pBests, float *gBest, float r1, 
                                     float r2)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // avoid an out of bound for the array 
    if(i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS)
        return;

    //float rp = getRandomClamped();
    //float rg = getRandomClamped();
    
    float rp = r1; // random weight for personnal =>  computed from @getRandomClamped
    float rg = r2; // random weight for global =>  computed from @getRandomClamped


    // Mise à jour de velocities et positions
    velocities[i] = OMEGA * velocities[i] + 
                    c1 * rp * (pBests[i] - positions[i]) + 
                    c2 * rg * (gBest[i % NUM_OF_DIMENSIONS] - positions[i]);

    // Update posisi particle
    //Mise à jour de la position de la particule courante
    //incrémentant la position de la particule courante avec la vitesse de la particule courante
    positions[i] += velocities[i];
}

/**
 * Runs on the GPU, called from the CPU or the GPU
*/
__global__ void kernelUpdatePBest(float *positions, float *pBests, float* gBest)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0)
        return;

    for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
    {
        tempParticle1[j] = positions[i + j];
        tempParticle2[j] = pBests[i + j];
    }

    if (fitness_function(tempParticle1) < fitness_function(tempParticle2))
    {
        for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
            pBests[i + k] = positions[i + k];
    }
}


extern "C" void cuda_pso(float *positions, float *velocities, float *pBests, float *gBest)
{

    int size = NUM_OF_PARTICLES * NUM_OF_DIMENSIONS;
    
    // declare all the arrays on the device
    float *devPos;
    float *devVel;
    float *devPBest;
    float *devGBest;
    
    float temp[NUM_OF_DIMENSIONS];
        
    // Memory allocation
    hipMalloc((void**)&devPos, sizeof(float) * size);
    hipMalloc((void**)&devVel, sizeof(float) * size);
    hipMalloc((void**)&devPBest, sizeof(float) * size);
    hipMalloc((void**)&devGBest, sizeof(float) * NUM_OF_DIMENSIONS);
    
    // Thread & Block number
    int threadsNum = 32;
    int blocksNum = ceil(size / threadsNum);
    
    // Copy particle datas from host to device
    /**
     * Copy in GPU memory the data from the host 
     * */
    hipMemcpy(devPos, positions, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devVel, velocities, sizeof(float) * size, 
               hipMemcpyHostToDevice);
    hipMemcpy(devPBest, pBests, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devGBest, gBest, sizeof(float) * NUM_OF_DIMENSIONS, 
               hipMemcpyHostToDevice);
    
    // PSO main function
    // MAX_ITER = 30000;

    for (int iter = 0; iter < MAX_ITER; iter++)
    {     

        kernelUpdateParticle<<<blocksNum, threadsNum>>>(devPos, devVel, 
                                                        devPBest, devGBest, 
                                                        getRandomClamped(), 
                                                        getRandomClamped());  

        kernelUpdatePBest<<<blocksNum, threadsNum>>>(devPos, devPBest, 
                                                     devGBest);
        
        hipMemcpy(pBests, devPBest, 
                   sizeof(float) * NUM_OF_PARTICLES * NUM_OF_DIMENSIONS, 
                   hipMemcpyDeviceToHost);
        
        
        for(int i = 0; i < size; i += NUM_OF_DIMENSIONS)
        {
            for(int k = 0; k < NUM_OF_DIMENSIONS; k++) //ssB1 
                temp[k] = pBests[i + k];
        
            if (host_fitness_function(temp) < host_fitness_function(gBest))
            {
                for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
                    gBest[k] = temp[k];
            }   
        }
        
        hipMemcpy(devGBest, gBest, sizeof(float) * NUM_OF_DIMENSIONS, 
                   hipMemcpyHostToDevice);
    }
    
    hipMemcpy(positions, devPos, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(velocities, devVel, sizeof(float) * size, 
               hipMemcpyDeviceToHost);
    hipMemcpy(pBests, devPBest, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(gBest, devGBest, sizeof(float) * NUM_OF_DIMENSIONS, 
               hipMemcpyDeviceToHost); 
    
    
    // cleanup
    hipFree(devPos);
    hipFree(devVel);
    hipFree(devPBest);
    hipFree(devGBest);
}

